#include "hip/hip_runtime.h"

//==============================================================================================//

#include <hip/hip_runtime.h> 
#include "../Utils/cudaUtil.h"
#include "../Utils/cuda_SimpleMatrixUtil.h"
#include "../Utils/RendererUtil.h"
#include "CUDABasedRasterizationGradInput.h"
#include "../Utils/CameraUtil.h"
#include "../Utils/IndexHelper.h"

//==============================================================================================//

/*
Initializes camera data
*/
__global__ void initializeCamerasGradDevice(CUDABasedRasterizationGradInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < 1)
	{
		for (int idc = 0; idc < input.numberOfCameras; idc++)
		{
			float4x4 h_intrinsics;
			float4x4 h_extrinsics;

			h_extrinsics.setIdentity();
			h_intrinsics.setIdentity();

			for (int row = 0; row < 3; row++)
			{
				h_intrinsics(row, 0) = input.d_cameraIntrinsics[3 * idc + row].x;
				h_intrinsics(row, 1) = input.d_cameraIntrinsics[3 * idc + row].y;
				h_intrinsics(row, 2) = input.d_cameraIntrinsics[3 * idc + row].z;
				h_intrinsics(row, 3) = 0.f;

				h_extrinsics(row, 0) = input.d_cameraExtrinsics[3 * idc + row].x;
				h_extrinsics(row, 1) = input.d_cameraExtrinsics[3 * idc + row].y;
				h_extrinsics(row, 2) = input.d_cameraExtrinsics[3 * idc + row].z;
				h_extrinsics(row, 3) = input.d_cameraExtrinsics[3 * idc + row].w;
			}

			float4x4 h_inExtrinsics = h_extrinsics.getInverse();
			float4x4 h_invProjection = (h_intrinsics * h_extrinsics).getInverse();

			for (int row = 0; row < 4; row++)
			{
				input.d_inverseExtrinsics[4 * idc + row].x = h_inExtrinsics(row, 0);
				input.d_inverseExtrinsics[4 * idc + row].y = h_inExtrinsics(row, 1);
				input.d_inverseExtrinsics[4 * idc + row].z = h_inExtrinsics(row, 2);
				input.d_inverseExtrinsics[4 * idc + row].w = h_inExtrinsics(row, 3);

				input.d_inverseProjection[4 * idc + row].x = h_invProjection(row, 0);
				input.d_inverseProjection[4 * idc + row].y = h_invProjection(row, 1);
				input.d_inverseProjection[4 * idc + row].z = h_invProjection(row, 2);
				input.d_inverseProjection[4 * idc + row].w = h_invProjection(row, 3);
			}
		}
	}
}

//==============================================================================================//

/*
Initialize gradients for lighting 
*/
__global__ void initBuffersGradDevice2(CUDABasedRasterizationGradInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input.numberOfCameras * 27)
	{
		input.d_shCoeffGrad[idx] = 0.f;
	}
}

//==============================================================================================//

/*
Initialize gradients for texture
*/
__global__ void initBuffersGradDevice1(CUDABasedRasterizationGradInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input.texHeight * input.texWidth)
	{
		input.d_textureGrad[idx] = make_float3(0.f,0.f,0.f);
	}
}

//==============================================================================================//

/*
Initialize gradients for mesh pos and color
*/
__global__ void initBuffersGradDevice0(CUDABasedRasterizationGradInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input.N)
	{
		input.d_vertexPosGrad[idx]	 = make_float3(0.f, 0.f, 0.f);
		input.d_vertexColorGrad[idx] = make_float3(0.f, 0.f, 0.f);
	}
}

//==============================================================================================//

/*
Get gradients for vertex color buffer
*/
__global__ void renderBuffersGradDevice(CUDABasedRasterizationGradInput input)
{
	const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx < input.numberOfCameras * input.w * input.h)
	{
		////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////
		//INDEXING
		////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////

		int3 index = index1DTo3D(input.numberOfCameras, input.h, input.w, idx);
		int idc = index.x;
		int idh = index.y;
		int idw = index.z;
		int idf = input.d_faceIDBuffer[idx];

		//still no face found
		if (idf == -1)
		{
			return;
		}

		////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////
		//INIT
		////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////

		float3 o = make_float3(0.f, 0.f, 0.f);
		float3 d = make_float3(0.f, 0.f, 0.f);
		float2 pixelPos = make_float2(idw + 0.5f, idh + 0.5f);
		getRayCuda2(pixelPos, o, d, input.d_inverseExtrinsics + idc * 4, input.d_inverseProjection + idc * 4);

		float2 bccTmp	= input.d_barycentricCoordinatesBuffer[idx];
		float3 bcc		= make_float3(bccTmp.x, bccTmp.y, 1.f - bccTmp.x - bccTmp.y);

		int3   faceVerticesIds  = input.d_facesVertex[idf];
		const float* shCoeff	= input.d_shCoeff + idc * 27;

		float3 vertexPos0 = input.d_vertices[faceVerticesIds.x];
		float3 vertexPos1 = input.d_vertices[faceVerticesIds.y];
		float3 vertexPos2 = input.d_vertices[faceVerticesIds.z];
		float3 vertexCol0 = input.d_vertexColor[faceVerticesIds.x];
		float3 vertexCol1 = input.d_vertexColor[faceVerticesIds.y];
		float3 vertexCol2 = input.d_vertexColor[faceVerticesIds.z];
		float3 vertexNor0 = input.d_vertexNormal[idc*input.N + faceVerticesIds.x];
		float3 vertexNor1 = input.d_vertexNormal[idc*input.N + faceVerticesIds.y];
		float3 vertexNor2 = input.d_vertexNormal[idc*input.N + faceVerticesIds.z];
		float2 texCoord0  = make_float2(input.d_textureCoordinates[idf * 3 * 2 + 0 * 2 + 0], 1.f - input.d_textureCoordinates[idf * 3 * 2 + 0 * 2 + 1]);
		float2 texCoord1  = make_float2(input.d_textureCoordinates[idf * 3 * 2 + 1 * 2 + 0], 1.f - input.d_textureCoordinates[idf * 3 * 2 + 1 * 2 + 1]);
		float2 texCoord2  = make_float2(input.d_textureCoordinates[idf * 3 * 2 + 2 * 2 + 0], 1.f - input.d_textureCoordinates[idf * 3 * 2 + 2 * 2 + 1]);

		float3 fragmentPosition = bcc.x * vertexPos0 + bcc.y * vertexPos1 + bcc.z * vertexPos2;

		float3 pixNormUn	= bcc.x * vertexNor0 + bcc.y * vertexNor1 + bcc.z * vertexNor2;
		float  pixNormVal	= sqrtf(pixNormUn.x*pixNormUn.x + pixNormUn.y*pixNormUn.y + pixNormUn.z*pixNormUn.z);
		float3 pixNorm		= pixNormUn / pixNormVal;

		bool flippedNormal = false;
		if (dot(pixNorm, d) > 0.f)
		{
			pixNorm = -pixNorm;
			flippedNormal = true;
		}

		float2 finalTexCoord = make_float2(0.f, 0.f);
		if (input.albedoMode == AlbedoMode::Textured)
		{
			finalTexCoord = texCoord0* bcc.x + texCoord1* bcc.y + texCoord2* bcc.z;
			finalTexCoord.x = finalTexCoord.x * input.texWidth;
			finalTexCoord.y = finalTexCoord.y * input.texHeight;
			finalTexCoord.x = fmaxf(finalTexCoord.x, 0);
			finalTexCoord.x = fminf(finalTexCoord.x, input.texWidth - 1);
			finalTexCoord.y = fmaxf(finalTexCoord.y, 0);
			finalTexCoord.y = fminf(finalTexCoord.y, input.texHeight - 1);
		}

		float3 pixLight = getIllum(pixNorm, shCoeff);
		mat3x3 JCoAl;

		if (input.shadingMode == ShadingMode::Shaded)
		{
			getJCoAl(JCoAl, pixLight);
		}
		else if (input.shadingMode == ShadingMode::Shadeless)
		{
			JCoAl.setIdentity();
		}

		mat3x3 JCoLi;
		float3 pixAlb = make_float3(0.f, 0.f, 0.f);
		if (input.albedoMode == AlbedoMode::VertexColor)
		{
			pixAlb = bcc.x * vertexCol0 + bcc.y * vertexCol1 + bcc.z * vertexCol2;
		}
		else if (input.albedoMode == AlbedoMode::Textured)
		{
			float U0 = finalTexCoord.x;
			float V0 = finalTexCoord.y;

			float  LU = int(finalTexCoord.x - 0.5f) + 0.5f;
			float  HU = int(finalTexCoord.x - 0.5f) + 1.5f;

			float  LV = int(finalTexCoord.y - 0.5f) + 0.5f;
			float  HV = int(finalTexCoord.y - 0.5f) + 1.5f;

			float3 colorLULV = make_float3(
				input.d_textureMap[3 * input.texWidth *(int)LV + 3 * (int)LU + 0],
				input.d_textureMap[3 * input.texWidth *(int)LV + 3 * (int)LU + 1],
				input.d_textureMap[3 * input.texWidth *(int)LV + 3 * (int)LU + 2]);

			float3 colorLUHV = make_float3(
				input.d_textureMap[3 * input.texWidth *(int)HV + 3 * (int)LU + 0],
				input.d_textureMap[3 * input.texWidth *(int)HV + 3 * (int)LU + 1],
				input.d_textureMap[3 * input.texWidth *(int)HV + 3 * (int)LU + 2]);

			float3 colorHULV = make_float3(
				input.d_textureMap[3 * input.texWidth *(int)LV + 3 * (int)HU + 0],
				input.d_textureMap[3 * input.texWidth *(int)LV + 3 * (int)HU + 1],
				input.d_textureMap[3 * input.texWidth *(int)LV + 3 * (int)HU + 2]);

			float3 colorHUHV = make_float3(
				input.d_textureMap[3 * input.texWidth *(int)HV + 3 * (int)HU + 0],
				input.d_textureMap[3 * input.texWidth *(int)HV + 3 * (int)HU + 1],
				input.d_textureMap[3 * input.texWidth *(int)HV + 3 * (int)HU + 2]);

			pixAlb = (V0 - LV) * (((U0 - LU) * colorLULV) + ((HU - U0) * colorHULV)) +
				(HV - V0) * (((U0 - LU) * colorLUHV) + ((HU - U0) * colorHUHV));
		}

		getJCoLi(JCoLi, pixAlb);

		////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////
		//VERTEX COLOR AND TEXTURE GRAD
		////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////

		mat1x3 GVCBVertexColor;
		GVCBVertexColor(0, 0) = input.d_renderBufferGrad[idx].x;
		GVCBVertexColor(0, 1) = input.d_renderBufferGrad[idx].y;
		GVCBVertexColor(0, 2) = input.d_renderBufferGrad[idx].z;

		if (input.albedoMode == AlbedoMode::VertexColor)
		{
			mat3x9 JAlVc;
			getJAlVc(JAlVc, bcc);

			mat1x9 gradVerCol = GVCBVertexColor * JCoAl * JAlVc;

			addGradients9I(gradVerCol.getTranspose(), input.d_vertexColorGrad, faceVerticesIds);
		}
		else if (input.albedoMode == AlbedoMode::Textured)
		{
			if (!flippedNormal)
			{
				mat1x3 gradTexColor = GVCBVertexColor * JCoAl;

				float  LU = int(finalTexCoord.x - 0.5f) + 0.5f;
				float  HU = int(finalTexCoord.x - 0.5f) + 1.5f;

				float  LV = int(finalTexCoord.y - 0.5f) + 0.5f;
				float  HV = int(finalTexCoord.y - 0.5f) + 1.5f;

				float U0 = finalTexCoord.x;
				float V0 = finalTexCoord.y;

				float weighting = 1.f;

				float weightLULV = (V0 - LV) * (U0 - LU);
				/*atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, LV, LU)].x, gradTexColor(0, 0) * weightLULV);
				atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, LV, LU)].y, gradTexColor(0, 1) * weightLULV);
				atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, LV, LU)].z, gradTexColor(0, 2) * weightLULV);

				float weightLUHV = (HV - V0) * (U0 - LU);
				atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, HV, LU)].x, gradTexColor(0, 0) * weightLUHV);
				atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, HV, LU)].y, gradTexColor(0, 1) * weightLUHV);
				atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, HV, LU)].z, gradTexColor(0, 2) * weightLUHV);

				float weightHULV = (V0 - LV) * (HU - U0);
				atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, LV, HU)].x, gradTexColor(0, 0) * weightHULV);
				atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, LV, HU)].y, gradTexColor(0, 1) * weightHULV);
				atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, LV, HU)].z, gradTexColor(0, 2) * weightHULV);

				float weightHUHV = (HV - V0) * (HU - U0);
				atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, HV, HU)].x, gradTexColor(0, 0) * weightHUHV);
				atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, HV, HU)].y, gradTexColor(0, 1) * weightHUHV);
				atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, HV, HU)].z, gradTexColor(0, 2) * weightHUHV);*/

				//printf("%f", weightLULV + weightLUHV + weightHULV + weightHUHV);

				atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, LV, LU)].x,  gradTexColor(0, 0) );
				atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, LV, LU)].y,  gradTexColor(0, 1) );
				atomicAdd(&input.d_textureGrad[index2DTo1D(input.texHeight, input.texWidth, LV, LU)].z,  gradTexColor(0, 2) );
			}
		}
		else
		{
			printf("Unsupported color mode in renderer gradient! \n");
		}
		
		////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////
		//LIGHTING GRAD
		////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////

		mat1x3 GVCBLight;
		GVCBLight(0, 0) = input.d_renderBufferGrad[idx].x;
		GVCBLight(0, 1) = input.d_renderBufferGrad[idx].y;
		GVCBLight(0, 2) = input.d_renderBufferGrad[idx].z;

		mat3x9 JLiGmR;
		getJLiGm(JLiGmR, 0, pixNorm);
		mat3x9 JLiGmG;
		getJLiGm(JLiGmG, 1, pixNorm);
		mat3x9 JLiGmB;
		getJLiGm(JLiGmB, 2, pixNorm);

		mat1x9 gradSHCoeffR;
		mat1x9 gradSHCoeffG;
		mat1x9 gradSHCoeffB;

		if (input.shadingMode == ShadingMode::Shaded)
		{
			gradSHCoeffR = GVCBLight * JCoLi * JLiGmR;
			gradSHCoeffG = GVCBLight * JCoLi * JLiGmG;
			gradSHCoeffB = GVCBLight * JCoLi * JLiGmB;
		}
		else if (input.shadingMode == ShadingMode::Shadeless)
		{
			gradSHCoeffR.setZero();
			gradSHCoeffG.setZero();
			gradSHCoeffB.setZero();
		}

		addGradients9(gradSHCoeffR, &input.d_shCoeffGrad[idc * 27]);
		addGradients9(gradSHCoeffG, &input.d_shCoeffGrad[idc * 27 + 9]);
		addGradients9(gradSHCoeffB, &input.d_shCoeffGrad[idc * 27 + 18]);

		////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////
		//VERTEX POS GRAD
		////////////////////////////////////////////////////////////////////////
		////////////////////////////////////////////////////////////////////////

		mat1x3 GVCBPosition;
		GVCBPosition(0, 0) = input.d_renderBufferGrad[idx].x;
		GVCBPosition(0, 1) = input.d_renderBufferGrad[idx].y;
		GVCBPosition(0, 2) = input.d_renderBufferGrad[idx].z;

		////////////////////////////////////////////////////////////////////////
		//data to model
		////////////////////////////////////////////////////////////////////////

		mat3x3 JNoNu;
		getJNoNu(JNoNu, pixNormUn, pixNormVal);

		mat3x3 JLiNo;
		getJLiNo(JLiNo, pixNorm, shCoeff);

		mat3x3 JAlBc;
		if (input.albedoMode == AlbedoMode::VertexColor)
		{
			getJAlBc(JAlBc, vertexCol0, vertexCol1, vertexCol2);
		}
		else if (input.albedoMode == AlbedoMode::Textured)
		{
			getJAlTexBc(JAlBc, input.d_textureMap, finalTexCoord, texCoord0, texCoord1, texCoord2, input.texWidth, input.texHeight, input.textureFilterSize);
		}
		else if (input.albedoMode == AlbedoMode::ForegroundMask)
		{
			getJAlBc(JAlBc, vertexCol0, vertexCol1, vertexCol2);
		}

		mat3x3 JNoBc;
		getJNoBc(JNoBc, vertexNor0, vertexNor1, vertexNor2);
		
		mat3x9 JBcVp;
		dJBCDVerpos(JBcVp, o, d, vertexPos0, vertexPos1, vertexPos2);

		mat1x9 gradVerPos;
		gradVerPos.setZero();
	
		gradVerPos = GVCBPosition * JCoAl * JAlBc * JBcVp;
	
		if (input.shadingMode == ShadingMode::Shaded)
		{
			gradVerPos = gradVerPos+ GVCBPosition * JCoLi * JLiNo * JNoNu * JNoBc * JBcVp ;
		}

		addGradients9I(gradVerPos.getTranspose(), input.d_vertexPosGrad, faceVerticesIds);

		////////////////////////////////////////////////////////////////////////
		//model to data
		////////////////////////////////////////////////////////////////////////

		// dT 3x2
		mat3x2 dT = imageGradient(((float3*)input.d_targetImage ) + idc * input.w * input.h , make_float2(idw, idh),input.w, input.h, input.imageFilterSize);
		 
		//dProj 2x3
		mat2x3 dProj;
		getJProjection(dProj, fragmentPosition, input.d_cameraIntrinsics + 3 * idc, input.d_cameraExtrinsics + 3 * idc);

		//dFrag 
		mat3x9 dFrag;
		dFrag.setZero();
		dFrag(0, 0) = bcc.x;
		dFrag(1, 1) = bcc.x;
		dFrag(2, 2) = bcc.x;

		dFrag(0, 3) = bcc.y;
		dFrag(1, 4) = bcc.y;
		dFrag(2, 5) = bcc.y;

		dFrag(0, 6) = bcc.z;
		dFrag(1, 7) = bcc.z;
		dFrag(2, 8) = bcc.z;

		mat1x9 model2DataGrad = -GVCBPosition * dT * dProj * dFrag;
		
		addGradients9I(model2DataGrad.getTranspose(), input.d_vertexPosGrad, faceVerticesIds);

		//////////////////////////////////////////////////////////////////////////////////

		if (input.shadingMode == ShadingMode::Shaded)
		{
			for (int i = 0; i < 3; i++)
			{
				mat3x3 JNuNvx;
				JNuNvx.setIdentity();
				int idv = -1;

				//
				if (i == 0)
				{
					idv = faceVerticesIds.x;
					JNuNvx = bcc.x * JNuNvx;
				}
				else if (i == 1)
				{
					idv = faceVerticesIds.y;
					JNuNvx = bcc.y * JNuNvx;
				}
				else
				{
					idv = faceVerticesIds.z;
					JNuNvx = bcc.z * JNuNvx;
				}

				int2 verFaceId = input.d_vertexFacesId[idv];
				for (int j = verFaceId.x; j < verFaceId.x + verFaceId.y; j++)
				{
					int faceId = input.d_vertexFaces[j];

					int3 v_index_inner = input.d_facesVertex[faceId];
					mat3x1 vi = (mat3x1)input.d_vertices[v_index_inner.x];
					mat3x1 vj = (mat3x1)input.d_vertices[v_index_inner.y];
					mat3x1 vk = (mat3x1)input.d_vertices[v_index_inner.z];

					mat3x3 J;

					// gradients vi
					getJ_vi(J, vk, vj, vi);
					mat1x3 gradVi = GVCBPosition * JCoLi * JLiNo * JNoNu * JNuNvx * J;
					addGradients(gradVi, &input.d_vertexPosGrad[v_index_inner.x]);

					// gradients vj
					getJ_vj(J, vk, vi);
					mat1x3 gradVj = GVCBPosition * JCoLi * JLiNo * JNoNu * JNuNvx * J;
					addGradients(gradVj, &input.d_vertexPosGrad[v_index_inner.y]);

					// gradients vk
					getJ_vk(J, vj, vi);
					mat1x3 gradVk = GVCBPosition * JCoLi * JLiNo * JNoNu * JNuNvx * J;
					addGradients(gradVk, &input.d_vertexPosGrad[v_index_inner.z]);
				}
			}
		}
	}
}

//==============================================================================================//

/*
Call to the devices for computing the gradients
*/
extern "C" void renderBuffersGradGPU(CUDABasedRasterizationGradInput& input)
{
	initializeCamerasGradDevice << < 1, 1 >> > (input);

	initBuffersGradDevice2    << < (input.numberOfCameras * 27 + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> >				(input);

	initBuffersGradDevice1    << < (input.texHeight * input.texWidth + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> >		(input);

	initBuffersGradDevice0    << < (input.N + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> >								(input);

	renderBuffersGradDevice   << < (input.numberOfCameras*input.w*input.h + THREADS_PER_BLOCK_CUDABASEDRASTERIZER - 1) / THREADS_PER_BLOCK_CUDABASEDRASTERIZER, THREADS_PER_BLOCK_CUDABASEDRASTERIZER >> >	(input);
}

//==============================================================================================//